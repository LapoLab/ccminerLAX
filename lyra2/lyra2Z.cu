#include "hip/hip_runtime.h"
#include "uint256.h"

extern "C" {
#include <sph/sph_blake.h>
#include "Lyra2Z.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void blake256_cpu_hash_112(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_112(uint32_t *pdata);

extern void lyra2Z_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2Z_cpu_init_sm2(int thr_id, uint32_t threads);
extern void lyra2Z_cpu_free(int thr_id);
extern uint32_t lyra2Z_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, bool gtx750ti);

extern void lyra2Z_setTarget(const void *ptarget);
extern uint32_t lyra2Z_getSecNonce(int thr_id, int num);

extern void lyra2Zz_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2Zz_cpu_init_sm2(int thr_id, uint32_t threads);
extern void lyra2Zz_cpu_free(int thr_id);
extern uint32_t lyra2Zz_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, bool gtx750ti);

extern void lyra2Zz_setTarget(const void *ptarget);
extern uint32_t lyra2Zz_getSecNonce(int thr_id, int num);
/*
extern "C" void lyra2Z_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hashA[8], hashB[8];
	sph_blake256_context ctx_blake;

	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	LYRA2Z(hashB, 32, hashA, 32, hashA, 32, 8, 8, 8);

	memcpy(state, hashB, 32);
}

extern "C" void lyra2Z_hash_112(void *state, const void *input)
{
	uint32_t _ALIGN(64) hashA[8], hashB[8];
	sph_blake256_context ctx_blake;

	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 112);
	sph_blake256_close(&ctx_blake, hashA);

	LYRA2Z(hashB, 32, hashA, 32, hashA, 32, 8, 8, 8);

	memcpy(state, hashB, 32);
}
*/

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;
static __thread bool gtx750ti = false;
static __thread size_t d_matrix_size = 0;

static size_t d_hash_size_bytes() 
{ 
	return (size_t)32 * throughput; 
}

static void maybe_init_thread_data(int thr_id, int dev_id, uint32_t max_nonce, uint32_t first_nonce)
{
	if (init[thr_id])
		return;

	hipSetDevice(dev_id);
	if (opt_cudaschedule == -1 && gpu_threads == 1) {
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		CUDA_LOG_ERROR();
	}

	cuda_get_arch(thr_id);
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 17 : 16;
	if (device_sm[dev_id] <= 500) intensity = 15;
	throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, dev_id);
	gtx750ti = (strstr(props.name, "750 Ti") != NULL);

	gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

	blake256_cpu_init(thr_id, throughput);

	if (device_sm[dev_id] >= 350)
	{
		size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
		d_matrix_size = matrix_sz;
		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
		lyra2Zz_cpu_init(thr_id, throughput, d_matrix[thr_id]);
	}
	else
		lyra2Zz_cpu_init_sm2(thr_id, throughput);

	CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], d_hash_size_bytes()));

	init[thr_id] = true;
}

extern "C" int scanhash_lyra2Z(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t _ALIGN(64) endiandata[20];
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		cuda_get_arch(thr_id);
		int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 17 : 16;
		if (device_sm[dev_id] <= 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);
		gtx750ti = (strstr(props.name, "750 Ti") != NULL);

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);

		if (device_sm[dev_id] >= 350)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			d_matrix_size = matrix_sz;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2Z_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		}
		else
			lyra2Z_cpu_init_sm2(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], d_hash_size_bytes()));

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	lyra2Z_setTarget(ptarget);

	do {
		int order = 0;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = lyra2Z_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], gtx750ti);

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			lyra2Z_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = lyra2Z_getSecNonce(thr_id, 1);
				work_set_target_ratio(work, vhash);
				pdata[19] = work->nonces[0] + 1;
				if (work->nonces[1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], work->nonces[1]);
					lyra2Z_hash(vhash, endiandata);
					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						work->valid_nonces++;
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)	gpulog(LOG_WARNING, thr_id,
					"result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0];
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

extern "C" int scanhash_lyra2Zz(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t _ALIGN(64) endiandata[28];
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		cuda_get_arch(thr_id);
		int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 17 : 16;
		if (device_sm[dev_id] <= 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);
		gtx750ti = (strstr(props.name, "750 Ti") != NULL);

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);

		if (device_sm[dev_id] >= 350)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			d_matrix_size = matrix_sz;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2Zz_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		} else {

			applog(LOG_ERR, "Lyra2Zz requires at least shader model 3.5 to work! Current shader model is %i. Exiting...",
				device_sm[dev_id]);

			return 0;
		}

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], d_hash_size_bytes()));

		init[thr_id] = true;
	}

//	for (int k=0; k < 28; k++) {
	//	be32enc(&endiandata[k], pdata[k]);
	//}
	
	//memcpy(endiandata, pdata, sizeof(endiandata));

	memcpy(&ptarget[0], work->target, sizeof(work->target));

	//blake256_cpu_setBlock_112(pdata);
	
	for (int k=0; k < 28; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	blake256_cpu_setBlock_112(endiandata);
	memcpy(endiandata, pdata, sizeof(endiandata));
	
	lyra2Zz_setTarget(ptarget);

	do {
		int order = 0;

		blake256_cpu_hash_112(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = lyra2Zz_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], gtx750ti);

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			//endiandata[19] = work->nonces[0];
			lyra2Z_hash_112(vhash, endiandata);
			
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = lyra2Z_getSecNonce(thr_id, 1);
				work_set_target_ratio(work, vhash);
				pdata[19] = work->nonces[0] + 1;
				if (work->nonces[1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], work->nonces[1]);
					lyra2Z_hash_112(vhash, endiandata);

					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						work->valid_nonces++;
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet) {
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				}
				pdata[19] = work->nonces[0];
				continue;
			}
		}

		if ((uint64_t) throughput + (uint64_t) pdata[19] >= (uint64_t) max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2Z(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	if (device_sm[dev_id] >= 350)
		hipFree(d_matrix[thr_id]);
	lyra2Z_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

static bool test_hash(int thr_id, uint32_t *input28)
{
	uint32_t start_n = input28[19];

	uint32_t adata[28];

	for (uint32_t i = 0; i < 28; ++i)
		be32enc(adata + i, input28[i]);

	blake256_cpu_init(thr_id, throughput);
	blake256_cpu_setBlock_112(adata);
	blake256_cpu_hash_112(thr_id, throughput, start_n, d_hash[thr_id], 0);

	uint256 target = uint256().SetCompact(input28[18]);
	lyra2Zz_setTarget(target.begin());
	lyra2Zz_cpu_hash_32(thr_id, throughput, start_n, d_hash[thr_id], gtx750ti);

	uint32_t correct = 0;

	for (size_t thread = 0; thread < throughput; ++thread) {				
		uint64_t gpu_state_hash[4];

		uint32_t out[8];
		be32enc(&input28[19], start_n + thread);
			
		lyra2Z_hash_112(out, input28);

		hipMemcpy(
			&gpu_state_hash[0], 
			d_matrix[thr_id] + 0 * throughput + (thread * 4), 
			sizeof(gpu_state_hash), 
			hipMemcpyDeviceToHost
		);

		if (memcmp(gpu_state_hash, out, sizeof(gpu_state_hash)) == 0) {
			correct++;
		}
	}

	return correct == throughput;
}

static bool niche_test(int thr_id)
{
	uint32_t testinput[28] = {
		0xa1e11c82,
		0xd2e52f4e,
		0x861eaf2d,
		0xe2ff5391,
		0x6e593b13,
		0xa698dc59,
		0xc1b59839,
		0x6e82926c,
		0x83bc8fcd,
		0x317ac43d,
		0x86c256f1,
		0x8b672c1b,
		0x81b7489e,
		0xa0d2a889,
		0x98862717,
		0x3a2d2244,
		0xeb5f6d94,
		0x8f7bb10e,
		0xdc14c194,
		0x00000000,
		0xc70194c4,
		0xeeb2112c,
		0x5e577c0e,
		0x0e39f176,
		0x6deddd3d,
		0xb3e09d44,
		0x624233d4,
		0x16f910b8
	};

	testinput[19] = 0;


	return test_hash(thr_id, testinput);
}

static bool large_test(int thr_id)
{
#ifdef _MSC_VER
	uint32_t correct = 0;
	const uint32_t num_tests = 1 << 8;

	LPCSTR cryptname = __FUNCTION__;
	HCRYPTPROV hCryptProv = NULL;

	if (!CryptAcquireContext(&hCryptProv, cryptname, NULL, PROV_RSA_FULL, CRYPT_NEWKEYSET)) {
		if (GetLastError() == NTE_EXISTS) {
			if (!CryptAcquireContext(&hCryptProv, cryptname, NULL, PROV_RSA_FULL, 0))
				goto ret_crypt_error;
		} else {
			goto ret_crypt_error;
		}
	}

	for (uint32_t i = 0; i < num_tests; ++i) {
		uint32_t adata[28];
		
		if (!CryptGenRandom(hCryptProv, sizeof(adata), (BYTE *)&adata[0])) {
			applog(
				LOG_WARNING, 
				__FUNCTION__ " Could not randomly generate buffer for iteration %u. Error: 0x%x\n", 
				i, 
				GetLastError()
			);
			continue;
		}

		adata[19] = 0;

		if (test_hash(thr_id, adata))
			correct++;
	}

	return correct == num_tests;

ret_crypt_error:
	applog(LOG_ERR, __FUNCTION__ " Could not get windows cryptography context - error returned: 0x%x\n", GetLastError());
		return false;
#endif

	return false;
}

extern "C" int lyra2Zz_test_hash(int thr_id, uint32_t *block_data)
{
	maybe_init_thread_data(thr_id, 0, UINT32_MAX, 0);
	return large_test(thr_id);
}


